#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_settings.h"
#include "cuda_helper.h"


__global__ void kernelComputeOpticFlow(int n,
                                       float* gpuFlowX,float* gpuFlowY,
                                       float** gpuEnergy,float* orientations, int orientationCnt){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        gpuFlowX[idx] = 0;
        gpuFlowY[idx] = 0;
        for(int i = 0; i  < orientationCnt; i++)
        {
            float energy = gpuEnergy[i][idx];
            gpuFlowX[idx] += energy*cos(orientations[i]);
            gpuFlowY[idx] += energy*sin(orientations[i]);
        }
    }
}

__host__ void cudaComputeOpticFlow(int sx, int sy,
                                  float* gpuFlowX, float* gpuFlowY,
                                  float** gpuArrGpuEnergy, float* gpuArrOrientations, int orientationCnt, hipStream_t stream)
{
    int n = sx*sy;
    long blocks = ceil((float)n/THREADS_PER_BLOCK);
    kernelComputeOpticFlow<<<blocks,THREADS_PER_BLOCK,0,stream>>>(
                         n,
                         gpuFlowX,gpuFlowY,
                         gpuArrGpuEnergy,gpuArrOrientations,orientationCnt);
}
