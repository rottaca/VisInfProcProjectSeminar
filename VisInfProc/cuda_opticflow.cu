#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_settings.h"
#include "cuda_helper.h"


__global__ void kernelComputeOpticFlow(int n,
                                       double* gpuFlowX,double* gpuFlowY,
                                       double** gpuEnergy,double* orientations, int orientationCnt){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        gpuFlowX[idx] = 0;
        gpuFlowY[idx] = 0;
        for(int i = 0; i  < orientationCnt; i++)
        {
            double energy = gpuEnergy[i][idx];
            gpuFlowX[idx] += energy*cos(orientations[i]);
            gpuFlowY[idx] += energy*sin(orientations[i]);
        }
    }
}

__host__ void cudaComputeOpticFlow(int sx, int sy,
                                  double* gpuFlowX, double* gpuFlowY,
                                  double** gpuArrGpuEnergy, double* gpuArrOrientations, int orientationCnt, hipStream_t stream)
{
    int n = sx*sy;
    long blocks = ceil((float)n/THREADS_PER_BLOCK);
    kernelComputeOpticFlow<<<blocks,THREADS_PER_BLOCK,0,stream>>>(
                         n,
                         gpuFlowX,gpuFlowY,
                         gpuArrGpuEnergy,gpuArrOrientations,orientationCnt);

}
