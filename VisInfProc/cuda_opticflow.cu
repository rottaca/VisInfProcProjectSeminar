#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_settings.h"
#include "cuda_helper.h"

__global__ void kernelComputeFlow(int n,
                                  float* gpuEnergy, float* gpuDir, float* gpuSpeed,
                                  float** gpuArrGpuEnergies,
                                  float* gpuArrOrientations, int orientationCnt,
                                  float* gpuArrSpeeds, int speedCnt,
                                  float minEnergy)
{
    int pixelIdx = threadIdx.x + blockIdx.x * blockDim.x;
    float flowX = 0;
    float flowY = 0;
    float energy = 0;
    float speed = 0;
    int energyIdx = 0;

    if(pixelIdx < n) {
        // Combine values from all orientations
        for(int i = 0; i < speedCnt; i++) {
            float localFlowX = 0, localFlowY = 0;
            float localEnergy = 0;
            for(int j = 0; j  < orientationCnt; j++) {
                localFlowX += gpuArrGpuEnergies[energyIdx][pixelIdx]*cos(gpuArrOrientations[j]);
                localFlowY += gpuArrGpuEnergies[energyIdx][pixelIdx]*sin(gpuArrOrientations[j]);
                energyIdx++;
            }
            localEnergy = sqrt(localFlowX*localFlowX+localFlowY*localFlowY);
            if(localEnergy > energy) {
                energy = localEnergy;
                flowX = localFlowX;
                flowY = localFlowY;
                speed = gpuArrSpeeds[i];
            }
        }

        gpuDir[pixelIdx] = 0;
        gpuEnergy[pixelIdx] = 0;
        gpuSpeed[pixelIdx] = 0;

        if(energy >= minEnergy) {
            gpuDir[pixelIdx] = atan2(flowY,flowX);
            gpuEnergy[pixelIdx] = energy;
            gpuSpeed[pixelIdx] = speed;
        }
    }
}
/**
 * @brief cudaComputeFlowEnergyAndDir Takes convolution buffers from all orientations
 *                                    and computes the flow energy and direction
 * @param sx
 * @param sy
 * @param gpuEnergy
 * @param gpuDir
 * @param gpuArrGpuEnergy Array of pointer to buffers
 * @param gpuArrOrientations
 * @param orientationCnt
 * @param speed
 * @param stream
 */
__host__ void cudaComputeFlow(int sx, int sy,
                              float* gpuEnergy, float* gpuDir, float* gpuSpeed,
                              float** gpuArrGpuEnergies,
                              float* gpuArrOrientations, int orientationCnt,
                              float* gpuArrSpeeds, int speedCnt,
                              float minEnergy,
                              hipStream_t stream)
{
    int n = sx*sy;
    size_t blocks = ceil((float)n/THREADS_PER_BLOCK);
    kernelComputeFlow<<<blocks,THREADS_PER_BLOCK,0,stream>>>(
        n,
        gpuEnergy,gpuDir,gpuSpeed,
        gpuArrGpuEnergies,
        gpuArrOrientations,orientationCnt,
        gpuArrSpeeds,speedCnt,
        minEnergy);
}

__global__ void kernelFlowToRGB(float* gpuEnergy, float* gpuDir, char *gpuImage,
                                int n,
                                float maxLength)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n) {
        float length = gpuEnergy[idx];
        float h = gpuDir[idx]/(2*M_PI)*360;
        if(h < 0)
            h+= 360;
        else if(h>= 360)
            h = 0;
        float s = length/maxLength;
        if(s > 1)
            s = 1;

        float v = 1;

        h /= 60.0f;
        int i = h;
        float ff = h-i;
        float p = v*(1.0f-s);
        float q = v*(1.0f - (s*ff));
        float t = v*(1.0f - (s* (1.0f-ff)));

        float r,g,b;
        switch(i) {
        case 0:
            r = v;
            g = t;
            b = p;
            break;
        case 1:
            r = q;
            g = v;
            b = p;
            break;
        case 2:
            r = p;
            g = v;
            b = t;
            break;
        case 3:
            r = p;
            g = q;
            b = v;
            break;
        case 4:
            r = t;
            g = p;
            b = v;
            break;
        case 5:
        default:
            r = v;
            g = p;
            b = q;
            break;
        }

        gpuImage[3*idx    ] = r*255;
        gpuImage[3*idx + 1] = g*255;
        gpuImage[3*idx + 2] = b*255;
    }
}

__host__ void cudaFlowToRGB(float* gpuEnergy, float* gpuDir, char *gpuImage,
                            int sx, int sy,
                            float maxLength, hipStream_t stream)
{
    int n = sx*sy;
    size_t blocks = ceil((float)n/THREADS_PER_BLOCK);
    kernelFlowToRGB<<<blocks,THREADS_PER_BLOCK,0,stream>>>(
        gpuEnergy,gpuDir,gpuImage,
        n,
        maxLength);
}
