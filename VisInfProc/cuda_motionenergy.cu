#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_settings.h"
#include "cuda_helper.h"
#include "datatypes.h"
#include <assert.h>

#define MAX_SHARED_GPU_EVENTS 256
__global__ void kernelProcessEventsBatchAsync(SimpleEvent* gpuEventList,int gpuEventListSize,
                            float* gpuFilter, int fsx, int fsy, int fsz,
                            float* gpuBuffer, int ringBufferIdx,
                            int bsx, int bsy, int bsz,
                            int fs_xy, int fn){

    // Calculate filter idx
    int filterPos = threadIdx.x + blockIdx.x * blockDim.x;
    // Idx valid
    if (filterPos < fn){
        float filterVal = gpuFilter[filterPos];
        // Compute x,y,z coodinates in buffer
        int fz = filterPos / fs_xy;
        int fxy = filterPos % fs_xy;
        int fy = fxy / fsx;
        int fx = fxy % fsx;

        // Convert buffer z index (flip z)
        int bz = ((ringBufferIdx + (fsz - 1) - fz ) % bsz);
        int bx_tmp = fsx / 2 - fx;
        int by_tmp = fsy / 2 - fy;
        int bPos_tmp = bz*bsy*bsx;

        // Per block shared memory
        __shared__ SimpleEvent gpuEventListShared[MAX_SHARED_GPU_EVENTS];
        int eventGroupCnt = ceil(gpuEventListSize/(float)MAX_SHARED_GPU_EVENTS);
        // Load events blockwise
        for(int eventGroupIdx = 0; eventGroupIdx<eventGroupCnt; eventGroupIdx++){
            int globalEventIdx = eventGroupIdx*MAX_SHARED_GPU_EVENTS+threadIdx.x/2;
            // The first MAX_SHARED_GPU_EVENTS threads copy the event data into shared memory
            if(threadIdx.x/2 < MAX_SHARED_GPU_EVENTS && globalEventIdx < gpuEventListSize){
                // even threads load x, odd threads load y
                if(threadIdx.x % 2 == 0){
                    gpuEventListShared[threadIdx.x/2].x = gpuEventList[globalEventIdx].x;
                }else{
                    gpuEventListShared[threadIdx.x/2].y = gpuEventList[globalEventIdx].y;
                }
            }
            // Synchronize
            __syncthreads();

            // Iterate over every event block in shared memory
            for(int localEventIdx = 0; localEventIdx < MAX_SHARED_GPU_EVENTS &&
                eventGroupIdx*MAX_SHARED_GPU_EVENTS+localEventIdx < gpuEventListSize; localEventIdx++){
                // Compute corresponding buffer coordinate (flip filter x,y)
                int bx = bx_tmp + gpuEventListShared[localEventIdx].x;
                int by = by_tmp + gpuEventListShared[localEventIdx].y;

                // Check for valid buffer position (filp buffer z)
                if(bx >= 0 && bx < bsx && by >= 0 && by < bsy){
                    int bufferPos = bPos_tmp + by*bsx + bx;
                    atomicAdd(gpuBuffer + bufferPos,filterVal);
                }
            }
        }
    }
}

__host__ void cudaProcessEventsBatchAsync(SimpleEvent* gpuEventList,int gpuEventListSize,
                                          float* gpuFilter, int fsx, int fsy, int fsz,
                                          float* gpuBuffer, int ringBufferIdx,
                                          int bsx, int bsy, int bsz,
                                          hipStream_t cudaStream)
{
    int fs_xy = fsx*fsy;
    int fn = fs_xy*fsz;
    long blocks = ceil((float)fn/THREADS_PER_BLOCK);
    kernelProcessEventsBatchAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(gpuEventList,gpuEventListSize,
                                                                             gpuFilter,fsx,fsy,fsz,
                                                                             gpuBuffer,ringBufferIdx,
                                                                             bsx,bsy,bsz,
                                                                             fs_xy,fn);
}

__global__ void kernelReadOpponentMotionEnergyAsync(float* gpuConvBufferl1,
                                                    float* gpuConvBufferl2,
                                                    float* gpuConvBufferr1,
                                                    float* gpuConvBufferr2,
                                                    int ringBufferIdx,
                                                    int bsx, int bsy, int bsz, int n,
                                                    float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm,
                                                    float* gpuEnergyBuffer){
    int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
    if(bufferPos < n){
        int bx,by,bz;
        int bxy = bufferpos / (bsx*bsy);
        bz = bufferPos % bsx*bsy;
        bx = bxy % bsx;
        by = bxy / bsx;

        // Offset in ringbuffer
        int bufferPosConv = bufferPos + ringBufferIdx*bsx*bsy;
        // Get answer from all 4 corresponding buffers and compute opponent motion energy
        // get all four filter responses and reset buffers
        float l1 = gpuConvBufferl1[bufferPosConv];
        gpuConvBufferl1[bufferPosConv] = 0;
        float l2 = gpuConvBufferl2[bufferPosConv];
        gpuConvBufferl2[bufferPosConv] = 0;
        float r1 = gpuConvBufferr1[bufferPosConv];
        gpuConvBufferr1[bufferPosConv] = 0;
        float r2 = gpuConvBufferr2[bufferPosConv];
        gpuConvBufferr2[bufferPosConv] = 0;

        // Compute opponent motion energy
        float energyR = sqrt(r1*r1+r2*r2);
        float energyL = sqrt(l1*l1+l2*l2);

        // Normalize energy
//        q_i = 0;
//        for(int y = -1; y <= 1; y++){
//            int by_ = by + y;
//            if(by_ < 0 || by_ >= bsy)
//                continue;
//            for(int x = -1; x <= 1; x++){
//                int bx_ = bx + x;
//                if(bx_ < 0 || bx_ >= bsx)
//                    continue;

//            }
//        }

        gpuEnergyBuffer[bufferPos] = energyR - energyL;
    }
}

__host__ void cudaReadOpponentMotionEnergyAsync(float* gpuConvBufferl1,
                                                float* gpuConvBufferl2,
                                                float* gpuConvBufferr1,
                                                float* gpuConvBufferr2,
                                                int ringBufferIdx,
                                                int bsx, int bsy, int bsz,
                                                float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm,
                                                float* gpuEnergyBuffer,
                                                hipStream_t cudaStream)
{
    int n = bsx*bsy;
    long blocks = ceil((float)n/THREADS_PER_BLOCK);
    kernelReadOpponentMotionEnergyAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(gpuConvBufferl1,
                                                                                   gpuConvBufferl2,
                                                                                   gpuConvBufferr1,
                                                                                   gpuConvBufferr2,
                                                                                   ringBufferIdx,bsx,bsy,bsz,n,
                                                                                   alphaPNorm,alphaQNorm,betaNorm,sigmaNorm,
                                                                                   gpuEnergyBuffer);
}
