#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_settings.h"
#include "cuda_helper.h"
#include "datatypes.h"
#include <assert.h>

__global__ void kernelProcessEventsBatchAsync(uint8_t* gpuEventsX,uint8_t* gpuEventsY,int gpuEventListSize,
        float* gpuFilter, int fsx, int fsy, int fsz,
        float* gpuBuffer, int ringBufferIdx,
        int bsx, int bsy, int bsz,
        int fs_xy, int fn)
{

    // Calculate filter idx
    int filterPos = threadIdx.x + blockIdx.x * blockDim.x;

    // Per block shared memory
    __shared__ uint8_t gpuEventListSharedX[MAX_SHARED_GPU_EVENTS];
    __shared__ uint8_t gpuEventListSharedY[MAX_SHARED_GPU_EVENTS];
    // How many runs do we need to process all events
    int processingRuns = ceilf((float)gpuEventListSize/MAX_SHARED_GPU_EVENTS);
    // Events for each thread to read
    int eventReadsPerThread = ceilf((float)MAX_SHARED_GPU_EVENTS/blockDim.x);
    // Offset n global event buffer
    int globalEventIdx = threadIdx.x;

    // Idx valid
    if (filterPos < fn) {
        // Read filter coefficient from global memory
        float filterVal = gpuFilter[filterPos];
        // Compute x,y,z coodinates in buffer
        int fz = filterPos / fs_xy;
        int fxy = filterPos % fs_xy;
        int fy = fxy / fsx;
        int fx = fxy % fsx;

        // Convert buffer coordinates (mirror all axes -> convolution instead of correlation)
        // Origin for mirroring is x = w/2, y = h/2, z = 0
        int bz = ((ringBufferIdx + (fsz - 1) - fz ) % bsz);
        int bx_tmp = fsx / 2 - fx;
        int by_tmp = fsy / 2 - fy;
        int bPos_tmp = bz*bsy*bsx;

        int sharedEventCnt = MAX_SHARED_GPU_EVENTS;
        // Iterate over event list in blocks, stored in shared memory
        for(int runIdx = 0; runIdx<processingRuns; runIdx++) {
            // Last run ? Compute size of shared event list
            if(runIdx+1 == processingRuns) {
                sharedEventCnt = gpuEventListSize % MAX_SHARED_GPU_EVENTS;
            }
            // Compute index in shared memory
            int localEventIdx = threadIdx.x;

            // Fill the shared memory either with MAX_SHARED_GPU_EVENTS
            // or use each thread mutlible times
            for(int i = 0; i < eventReadsPerThread; i++) {
                // Valid indices
                if(localEventIdx >= sharedEventCnt)
                    break;
                // Load event into shared memory by using one thread per event
                gpuEventListSharedX[localEventIdx] = gpuEventsX[globalEventIdx];
                gpuEventListSharedY[localEventIdx] = gpuEventsY[globalEventIdx];

                // Goto next event for which this thread is responsible
                localEventIdx += blockDim.x;
                globalEventIdx += blockDim.x;
            }

            // Synchronize threads and wait until shared memory is filled
            // TODO: Deadlock possible?
            // At least one thread in each warp should hit that barrier to continue!
            // Bad relationship between shared event list size and block size could cause problems ?!
            __syncthreads();

            // Iterate over every event block in shared memory
            for(localEventIdx = 0; localEventIdx < sharedEventCnt; localEventIdx++) {
                // Compute corresponding buffer coordinate
                int bx = bx_tmp + gpuEventListSharedX[localEventIdx];
                int by = by_tmp + gpuEventListSharedY[localEventIdx];

                // Check for valid buffer position (filp buffer z)
                if(bx >= 0 && bx < bsx && by >= 0 && by < bsy) {
                    int bufferPos = bPos_tmp + by*bsx + bx;
                    // Add each filter coefficient to the global buffer
                    atomicAdd(gpuBuffer + bufferPos,filterVal);
                }
            }
        }
    }
}
/**
 * @brief cudaProcessEventsBatchAsync Processes a gpu event list with a given
 *                                    filter and stores the result in the given buffer
 * @param gpuEventsX
 * @param gpuEventsY
 * @param gpuEventListSize
 * @param gpuFilter
 * @param fsx
 * @param fsy
 * @param fsz
 * @param gpuBuffer
 * @param ringBufferIdx
 * @param bsx
 * @param bsy
 * @param bsz
 * @param cudaStream
 */
__host__ void cudaProcessEventsBatchAsync(uint8_t* gpuEventsX,uint8_t* gpuEventsY,int gpuEventListSize,
        float* gpuFilter, int fsx, int fsy, int fsz,
        float* gpuBuffer, int ringBufferIdx,
        int bsx, int bsy, int bsz,
        hipStream_t cudaStream)
{
    int fs_xy = fsx*fsy;
    int fn = fs_xy*fsz;
    size_t blocks = ceilf((float)fn/THREADS_PER_BLOCK);
    kernelProcessEventsBatchAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(gpuEventsX,gpuEventsY,gpuEventListSize,
            gpuFilter,fsx,fsy,fsz,
            gpuBuffer,ringBufferIdx,
            bsx,bsy,bsz,
            fs_xy,fn);
}

__global__ void kernelReadMotionEnergyAsync(float* gpuConvBufferl1,
        float* gpuConvBufferl2,
        int ringBufferIdx,
        int bsx, int bsy, int n,
        float* gpuEnergyBuffer)
{
    int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
    if(bufferPos < n) {
        // Offset in ringbuffer
        int bufferPosConv = bufferPos + ringBufferIdx*bsx*bsy;
        // Get answer from two corresponding buffers and compute motion energy
        float l1 = gpuConvBufferl1[bufferPosConv];
        float l2 = gpuConvBufferl2[bufferPosConv];

        // Compute motion energy
        gpuEnergyBuffer[bufferPos] = sqrt(l1*l1+l2*l2);
    }
}
/**
 * @brief cudaReadMotionEnergyAsync Reads the motionenergy from the two
 *                                  corresponding convolution buffers
 *                                  and stores the energy in a gpu buffer.
 * @param gpuConvBufferl1
 * @param gpuConvBufferl2
 * @param ringBufferIdx
 * @param bsx
 * @param bsy
 * @param gpuEnergyBuffer
 * @param cudaStream
 */
__host__ void cudaReadMotionEnergyAsync(float* gpuConvBufferl1,
                                        float* gpuConvBufferl2,
                                        int ringBufferIdx,
                                        int bsx, int bsy,
                                        float* gpuEnergyBuffer,
                                        hipStream_t cudaStream)
{
    int n = bsx*bsy;
    size_t blocks = ceilf((float)n/THREADS_PER_BLOCK);
    kernelReadMotionEnergyAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(gpuConvBufferl1,
            gpuConvBufferl2,
            ringBufferIdx,bsx,bsy,n,
            gpuEnergyBuffer);
}


__global__ void kernelNormalizeMotionEnergyAsync(int bsx, int bsy, int n,
        float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm,
        float* gpuEnergyBuffer)
{
    int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
    float sigmaNorm2_2 = 2*sigmaNorm*sigmaNorm;
    if(bufferPos < n) {
        int bx,by;
        int bxy = bufferPos / (bsx*bsy);
        bx = bxy % bsx;
        by = bxy / bsx;
        // Read energy
        float I = gpuEnergyBuffer[bufferPos];
        float q_i = 0;
        // Normalize over 5x5 region
        for(int y = -2; y <= 2; y++) {
            int by_ = by + y;

            if(by_ < 0 || by_ >= bsy)
                continue;

            for(int x = -2; x <= 2; x++) {
                int bx_ = bx + x;

                if(bx_ < 0 || bx_ >= bsx ||
                        (bx == bx_ && by == by_))
                    continue;
                // TODO
                // Each thread computes the same
                float gaus = 1/(sigmaNorm2_2*M_PI)* exp(-(bx_*bx_ + by_*by_)/sigmaNorm2_2);
                // TODO Use shared memory to avoid extra global memory access
                q_i += gpuEnergyBuffer[by_*bsx+bx_]*gaus;
            }
        }
        q_i /= alphaQNorm;

        // Compute p_i
        float p_i = (I*betaNorm)/(alphaPNorm + I + q_i);

        // Use normalized value
        gpuEnergyBuffer[bufferPos] = p_i;
    }
}
/**
 * @brief cudaNormalizeMotionEnergyAsync Normalizes the motion energy
 *                                       inplace.
 * @param bsx
 * @param bsy
 * @param alphaPNorm
 * @param alphaQNorm
 * @param betaNorm
 * @param sigmaNorm
 * @param gpuEnergyBuffer
 * @param cudaStream
 */
__host__ void cudaNormalizeMotionEnergyAsync(int bsx, int bsy,
        float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm,
        float* gpuEnergyBuffer,
        hipStream_t cudaStream)
{
    int n = bsx*bsy;
    size_t blocks = ceilf((float)n/THREADS_PER_BLOCK);
    kernelNormalizeMotionEnergyAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(bsx,bsy,n,
            alphaPNorm,alphaQNorm,betaNorm,sigmaNorm,
            gpuEnergyBuffer);
}
