#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_settings.h"
#include "cuda_helper.h"
#include "datatypes.h"
#include <assert.h>

__global__ void kernelProcessEventsBatchAsync(uint8_t* gpuEventsX,uint8_t* gpuEventsY,int gpuEventListSize,
        float* gpuFilter, int fsx, int fsy, int fsz,
        float* gpuBuffer, int ringBufferIdx,
        int bsx, int bsy, int bsz,
        int fs_xy, int fn)
{

    // Calculate filter idx
    int filterPos = threadIdx.x + blockIdx.x * blockDim.x;
    // Idx valid
    if (filterPos < fn) {
        float filterVal = gpuFilter[filterPos];
        // Compute x,y,z coodinates in buffer
        int fz = filterPos / fs_xy;
        int fxy = filterPos % fs_xy;
        int fy = fxy / fsx;
        int fx = fxy % fsx;

        // Convert buffer z index (flip z)
        int bz = ((ringBufferIdx + (fsz - 1) - fz ) % bsz);
        int bx_tmp = fsx / 2 - fx;
        int by_tmp = fsy / 2 - fy;
        int bPos_tmp = bz*bsy*bsx;

        // Per block shared memory
        __shared__ uint8_t gpuEventListSharedX[MAX_SHARED_GPU_EVENTS];
        __shared__ uint8_t gpuEventListSharedY[MAX_SHARED_GPU_EVENTS];
        // How many runs do we need to process all events
        int processingRuns = ceil((float)gpuEventListSize/MAX_SHARED_GPU_EVENTS);
        // Events for each thread to read
        int eventReadsPerThread = ceil((float)MAX_SHARED_GPU_EVENTS/blockDim.x);
        // Load events blockwise
        for(int runIdx = 0; runIdx<processingRuns; runIdx++) {
            // Fill the shared memory either with MAX_SHARED_GPU_EVENTS
            // or use each thread mutlible times
            for(int i = 0; i < eventReadsPerThread; i++) {
                // Compute index in shared memory
                int localEventIdx = i*blockDim.x+threadIdx.x;
                // Compute index in global event memory
                int globalEventIdx = runIdx*MAX_SHARED_GPU_EVENTS+localEventIdx;
                // Valid indices
                if(globalEventIdx < gpuEventListSize && localEventIdx < MAX_SHARED_GPU_EVENTS) {
                    gpuEventListSharedX[localEventIdx] = gpuEventsX[globalEventIdx];
                    gpuEventListSharedY[localEventIdx] = gpuEventsY[globalEventIdx];
                }
            }

            // Synchronize threads and wait until shared memory is filled
            __syncthreads();

            // Iterate over every event block in shared memory
            for(int localEventIdx = 0; localEventIdx < MAX_SHARED_GPU_EVENTS &&
                    runIdx*MAX_SHARED_GPU_EVENTS+localEventIdx < gpuEventListSize; localEventIdx++) {
                // Compute corresponding buffer coordinate (flip filter x,y)
                int bx = bx_tmp + gpuEventListSharedX[localEventIdx];
                int by = by_tmp + gpuEventListSharedY[localEventIdx];

                // Check for valid buffer position (filp buffer z)
                if(bx >= 0 && bx < bsx && by >= 0 && by < bsy) {
                    int bufferPos = bPos_tmp + by*bsx + bx;
                    // Add each filter coefficient to the global buffer
                    atomicAdd(gpuBuffer + bufferPos,filterVal);
                }
            }
        }
    }
}
/**
 * @brief cudaProcessEventsBatchAsync Processes a gpu event list with a given
 *                                    filter and stores the result in the given buffer
 * @param gpuEventsX
 * @param gpuEventsY
 * @param gpuEventListSize
 * @param gpuFilter
 * @param fsx
 * @param fsy
 * @param fsz
 * @param gpuBuffer
 * @param ringBufferIdx
 * @param bsx
 * @param bsy
 * @param bsz
 * @param cudaStream
 */
__host__ void cudaProcessEventsBatchAsync(uint8_t* gpuEventsX,uint8_t* gpuEventsY,int gpuEventListSize,
        float* gpuFilter, int fsx, int fsy, int fsz,
        float* gpuBuffer, int ringBufferIdx,
        int bsx, int bsy, int bsz,
        hipStream_t cudaStream)
{
    int fs_xy = fsx*fsy;
    int fn = fs_xy*fsz;
    size_t blocks = ceil((float)fn/THREADS_PER_BLOCK);
    kernelProcessEventsBatchAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(gpuEventsX,gpuEventsY,gpuEventListSize,
            gpuFilter,fsx,fsy,fsz,
            gpuBuffer,ringBufferIdx,
            bsx,bsy,bsz,
            fs_xy,fn);
}

__global__ void kernelReadMotionEnergyAsync(float* gpuConvBufferl1,
        float* gpuConvBufferl2,
        int ringBufferIdx,
        int bsx, int bsy, int n,
        float* gpuEnergyBuffer)
{
    int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
    if(bufferPos < n) {
        // Offset in ringbuffer
        int bufferPosConv = bufferPos + ringBufferIdx*bsx*bsy;
        // Get answer from two corresponding buffers and compute motion energy
        float l1 = gpuConvBufferl1[bufferPosConv];
        float l2 = gpuConvBufferl2[bufferPosConv];

        // Compute motion energy
        gpuEnergyBuffer[bufferPos] = sqrt(l1*l1+l2*l2);
    }
}
/**
 * @brief cudaReadMotionEnergyAsync Reads the motionenergy from the two
 *                                  corresponding convolution buffers
 *                                  and stores the energy in a gpu buffer.
 * @param gpuConvBufferl1
 * @param gpuConvBufferl2
 * @param ringBufferIdx
 * @param bsx
 * @param bsy
 * @param gpuEnergyBuffer
 * @param cudaStream
 */
__host__ void cudaReadMotionEnergyAsync(float* gpuConvBufferl1,
                                        float* gpuConvBufferl2,
                                        int ringBufferIdx,
                                        int bsx, int bsy,
                                        float* gpuEnergyBuffer,
                                        hipStream_t cudaStream)
{
    int n = bsx*bsy;
    size_t blocks = ceil((float)n/THREADS_PER_BLOCK);
    kernelReadMotionEnergyAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(gpuConvBufferl1,
            gpuConvBufferl2,
            ringBufferIdx,bsx,bsy,n,
            gpuEnergyBuffer);
}


__global__ void kernelNormalizeMotionEnergyAsync(int bsx, int bsy, int n,
        float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm,
        float* gpuEnergyBuffer)
{
    int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
    if(bufferPos < n) {
        int bx,by;
        int bxy = bufferPos / (bsx*bsy);
        bx = bxy % bsx;
        by = bxy / bsx;
        // Read energy
        float I = gpuEnergyBuffer[bufferPos];
        float q_i = 0;
        // Normalize over 5x5 region
        for(int y = -2; y <= 2; y++) {
            int by_ = by + y;

            if(by_ < 0 || by_ >= bsy)
                continue;

            for(int x = -2; x <= 2; x++) {
                int bx_ = bx + x;

                if(bx_ < 0 || bx_ >= bsx ||
                        (bx == bx_ && by == by_))
                    continue;
                // TODO
                // Each thread computes the same
                float gaus = 1/(2*sigmaNorm*sigmaNorm*M_PI)* exp(-(bx_*bx_ + by_*by_)/(2*sigmaNorm*sigmaNorm));
                // TODO Use shared memory to avoid extra global memory access
                q_i += gpuEnergyBuffer[by_*bsx+bx_]*gaus;
            }
        }
        q_i /= alphaQNorm;

        // Compute p_i
        float p_i = (I*betaNorm)/(alphaPNorm + I + q_i);

        // Use normalized value
        gpuEnergyBuffer[bufferPos] = p_i;
    }
}
/**
 * @brief cudaNormalizeMotionEnergyAsync Normalizes the motion energy
 *                                       inplace.
 * @param bsx
 * @param bsy
 * @param alphaPNorm
 * @param alphaQNorm
 * @param betaNorm
 * @param sigmaNorm
 * @param gpuEnergyBuffer
 * @param cudaStream
 */
__host__ void cudaNormalizeMotionEnergyAsync(int bsx, int bsy,
        float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm,
        float* gpuEnergyBuffer,
        hipStream_t cudaStream)
{
    int n = bsx*bsy;
    size_t blocks = ceil((float)n/THREADS_PER_BLOCK);
    kernelNormalizeMotionEnergyAsync<<<blocks,THREADS_PER_BLOCK,0,cudaStream>>>(bsx,bsy,n,
            alphaPNorm,alphaQNorm,betaNorm,sigmaNorm,
            gpuEnergyBuffer);
}
