#include "hip/hip_runtime.h"
#ifndef _CU_HELPER_
#define _CU_HELPER_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_settings.h"
#include "cuda_helper.h"



__host__ double* cudaCreateBuffer(long size){
    double* gpuPtr;
    gpuErrchk(hipMalloc(&gpuPtr,size*sizeof(double)));
    if(gpuPtr == 0)
    {
        fprintf(stderr,"Cuda malloc failed!\n");
        exit(-1);
    }
    return gpuPtr;
}

__host__ void cudaUploadBuffer(double* cpuBuffPtr, double * gpuBuffPtr,long size){
    gpuErrchk(hipMemcpy(gpuBuffPtr,cpuBuffPtr,size*sizeof(double),hipMemcpyHostToDevice));
}

__global__ void kernelSetBuffer(double* gpuBuffPtr, double v, long size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size)
        gpuBuffPtr[index] = v;
}

__host__ void cudaSetBuffer(double* gpuBuffPtr,double v, long size){
    kernelSetBuffer<<<size/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(gpuBuffPtr,v,size);
}

__host__ void cudaDownloadBuffer(double* gpuBuffPtr, double * cpuBuffPtr,long size){
    gpuErrchk(hipMemcpy(cpuBuffPtr,gpuBuffPtr,size*sizeof(double),hipMemcpyDeviceToHost));
}
__host__ void cudaFreeBuffer(double* gpuBuffPtr){
    gpuErrchk(hipFree(gpuBuffPtr));
}

#endif
