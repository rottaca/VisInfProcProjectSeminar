#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_settings.h"
#include "cuda_helper.h"

#include <iostream>

__host__ void* cudaCreateBuffer(long size){
    if(size == 0)
        return NULL;
    void* gpuPtr;
    gpuErrchk(hipMalloc(&gpuPtr,size));
    if(gpuPtr == 0)
    {
        fprintf(stderr,"Cuda malloc failed! Can't create buffer of size: %d\n",size);
        exit(-1);
    }
    return gpuPtr;
}
__host__ void cudaUploadBuffer(void* cpuBuffPtr, void* gpuBuffPtr,long size,hipStream_t stream){
    gpuErrchk(hipMemcpyAsync(gpuBuffPtr,cpuBuffPtr,size,hipMemcpyHostToDevice,stream));
}
__host__ void cudaDownloadBuffer(void* gpuBuffPtr, void * cpuBuffPtr,long size,hipStream_t stream){
    gpuErrchk(hipMemcpyAsync(cpuBuffPtr,gpuBuffPtr,size,hipMemcpyDeviceToHost,stream));
}
__host__ void cudaCopyBuffer(void* gpuBuffPtrDest, void * gpuBuffPtrSrc,long size,hipStream_t stream){
    gpuErrchk(hipMemcpyAsync(gpuBuffPtrDest,gpuBuffPtrSrc,size,hipMemcpyDeviceToDevice,stream));
}
__host__ void cudaFreeBuffer(void* gpuBuffPtr){
    gpuErrchk(hipFree(gpuBuffPtr));
}

__global__ void kernelSetDoubleBuffer(float* gpuBuffPtr, float v, long size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size)
        gpuBuffPtr[index] = v;
}

__host__ void cudaSetDoubleBuffer(float* gpuBuffPtr,float v, long size,hipStream_t stream){
    // Run through filter buffer
    long blocks = ceil((float)size/THREADS_PER_BLOCK);
    kernelSetDoubleBuffer<<<blocks,THREADS_PER_BLOCK,0,stream>>>(gpuBuffPtr,v,size);
}
