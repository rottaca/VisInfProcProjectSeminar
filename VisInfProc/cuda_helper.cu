#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_settings.h"
#include "cuda_helper.h"

#include <iostream>

__host__ void* cudaCreateBuffer(long size){
    void* gpuPtr;
    gpuErrchk(hipMalloc(&gpuPtr,size));
    if(gpuPtr == 0)
    {
        fprintf(stderr,"Cuda malloc failed!\n");
        exit(-1);
    }
    return gpuPtr;
}
__host__ void cudaUploadBuffer(void* cpuBuffPtr, void* gpuBuffPtr,long size){
    gpuErrchk(hipMemcpy(gpuBuffPtr,cpuBuffPtr,size,hipMemcpyHostToDevice));
}
__host__ void cudaDownloadBuffer(void* gpuBuffPtr, void * cpuBuffPtr,long size){
    gpuErrchk(hipMemcpy(cpuBuffPtr,gpuBuffPtr,size,hipMemcpyDeviceToHost));
}
__host__ void cudaCopyBuffer(void* gpuBuffPtrDest, void * gpuBuffPtrSrc,long size){
    gpuErrchk(hipMemcpy(gpuBuffPtrDest,gpuBuffPtrSrc,size,hipMemcpyDeviceToDevice));
}
__host__ void cudaFreeBuffer(void* gpuBuffPtr){
    gpuErrchk(hipFree(gpuBuffPtr));
}

__global__ void kernelSetDoubleBuffer(double* gpuBuffPtr, double v, long size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size)
        gpuBuffPtr[index] = v;
}

__host__ void cudaSetDoubleBuffer(double* gpuBuffPtr,double v, long size){
    // Run through filter buffer
    long blocks = ceil((float)size/THREADS_PER_BLOCK);
    kernelSetDoubleBuffer<<<blocks,THREADS_PER_BLOCK>>>(gpuBuffPtr,v,size);
}
