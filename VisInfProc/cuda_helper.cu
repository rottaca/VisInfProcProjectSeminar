#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_settings.h"
#include "cuda_helper.h"

#include <iostream>

__host__ void* cudaCreateBuffer(size_t size)
{
    if(size == 0)
        return NULL;
    void* gpuPtr;
    gpuErrchk(hipMalloc(&gpuPtr,size));
    if(gpuPtr == 0)
        {
            fprintf(stderr,"Cuda malloc failed! Can't create buffer of size: %ld\n",size);
            exit(-1);
        }
    return gpuPtr;
}
__host__ void cudaUploadBuffer(void* cpuBuffPtr, void* gpuBuffPtr, size_t size, hipStream_t stream)
{
    gpuErrchk(hipMemcpyAsync(gpuBuffPtr,cpuBuffPtr,size,hipMemcpyHostToDevice,stream));
}
__host__ void cudaDownloadBuffer(void* gpuBuffPtr, void * cpuBuffPtr,size_t size,hipStream_t stream)
{
    gpuErrchk(hipMemcpyAsync(cpuBuffPtr,gpuBuffPtr,size,hipMemcpyDeviceToHost,stream));
}
__host__ void cudaCopyBuffer(void* gpuBuffPtrDest, void * gpuBuffPtrSrc, size_t size, hipStream_t stream)
{
    gpuErrchk(hipMemcpyAsync(gpuBuffPtrDest,gpuBuffPtrSrc,size,hipMemcpyDeviceToDevice,stream));
}
__host__ void cudaFreeBuffer(void* gpuBuffPtr)
{
    gpuErrchk(hipFree(gpuBuffPtr));
}

__global__ void kernelSetDoubleBuffer(float* gpuBuffPtr, float v, size_t size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size)
        gpuBuffPtr[index] = v;
}

__host__ void cudaSetDoubleBuffer(float* gpuBuffPtr,float v, size_t size,hipStream_t stream)
{
    // Run through filter buffer
    size_t blocks = ceil((float)size/THREADS_PER_BLOCK);
    kernelSetDoubleBuffer<<<blocks,THREADS_PER_BLOCK,0,stream>>>(gpuBuffPtr,v,size);
}
